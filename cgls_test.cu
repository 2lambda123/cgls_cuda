#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <cmath>

#include "cgls.cuh"

// Define real type.
typedef double real_t;
#define csr2csc hipsparseDcsr2csc

// Generates random CSR matrix with entries in [-1, 1]. The matrix will have
// exactly nnz non-zeros. All arrays must be pre-allocated.
void CsrMatGen(int m, int n, int nnz, real_t *val, int *rptr, int *cind) {
  real_t kRandMax = static_cast<real_t>(RAND_MAX);
  real_t kM = static_cast<real_t>(m);
  real_t kN = static_cast<real_t>(n);

  int num = 0;
  for (int i = 0; i < m; ++i) {
    rptr[i] = num;
    for (int j = 0; j < n && num < nnz; ++j) {
      if (rand() / kRandMax * ((kM - i) * kN - j) < (nnz - num)) {
        val[num] = 2 * (rand() - kRandMax / 2) / kRandMax;
        cind[num] = j;
        num++;
      }
    }
  }
  rptr[m] = nnz;
}

// Test CGLS on square system of equations with known solution.
void test1() {
  // Initialize variables.
  real_t shift = 1;
  real_t tol = 1e-6;
  int maxit = 20;
  bool quiet = false;
  int m = 5;
  int n = 5;
  int nnz = 13;

  // Initialize data.
  real_t val_h[]  = { 1, -1, -3, -2,  5,  4,  6,  4, -4,  2,  7,  8, -5};
  int cind_h[]   = {0, 1, 3, 0, 1, 2, 3, 4, 0, 2, 3, 1, 4};
  int rptr_h[]   = {0, 3, 5, 8, 11, 13};
  real_t b_h[]    = {-2, -1,  0,  1,  2};
  real_t x_h[]    = {0,  0,  0,  0,  0};
  real_t x_star[] = { 0.461620337853983,  0.025458521291462, -0.509793131412600,
                      0.579159637092979, -0.350590484189795};

  // Transfer variables to device.
  real_t *val_d, *b_d, *x_d;
  int *cind_d, *rptr_d;

  hipMalloc(&val_d, (nnz + m + n) * sizeof(real_t));
  hipMalloc(&cind_d, (nnz + m + 1) * sizeof(int));
  b_d = val_d + nnz;
  x_d = b_d + m;
  rptr_d = cind_d + nnz;

  hipMemcpy(val_d, val_h, nnz * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, m * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(x_d, x_h, n * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(cind_d, cind_h, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(rptr_d, rptr_h, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

  // Solve.
  int flag = cgls::solve<real_t, cgls::CSR>(val_d, rptr_d, cind_d, m, n,
      nnz, b_d, x_d, shift, tol, maxit, quiet);

  // Retrieve solution.
  hipMemcpy(x_h, x_d, n * sizeof(real_t), hipMemcpyDeviceToHost);

  // Compute error and print.
  real_t err = 0;
  for (int i = 0; i < n; ++i)
    err += (x_h[i] - x_star[i]) * (x_h[i] - x_star[i]);
  err = std::sqrt(err);
  if (flag == 0 && err < tol)
    printf("Test1 Passed: Flag = %d, Error = %e\n", flag, err);
  else
    printf("Test1 Failed: Flag = %d, Error = %e\n", flag, err);

  // Free data.
  hipFree(val_d);
  hipFree(cind_d);
}

// Test CGLS on rectangular system of equations with known solution.
void test2() {
  // Initialize variables.
  real_t shift = 0.1;
  real_t tol = 1e-6;
  int maxit = 20;
  bool quiet = false;
  int m = 10;
  int n = 5;
  int nnz = 17;

  // Initialize data.
  real_t val_h[]  = { 0.503206792576615, -0.064342931468363,  0.273525398508089,
                     -0.876171296658172,  0.699498416627245,  0.006382734094307,
                     -0.872461490857631, -1.927164633937109, -1.655186057400025,
                      0.140300920195852,  0.745416695810262, -0.949513158012032,
                      0.753179647233809,  0.117556530400676, -1.458256332931324,
                     -0.742412119936071, -0.269214611464301};
  int cind_h[] = {0, 1, 2, 3, 2, 3, 3, 4, 0, 1, 0, 2, 3, 1, 4, 4, 0};
  int rptr_h[] = {0, 4, 6, 6, 8, 10, 13, 14, 15, 16, 17};
  real_t b_h[]    = { 1.340034585145723, -0.634242023306306, -0.213297722346186,
                     -0.129598039513105,  0.132020354623637,  0.078143427011308,
                      0.300482010299278, -0.688536305275490, -0.465698657933079,
                      0.074768275950993};
  real_t x_h[]    = {0, 0, 0, 0, 0};
  real_t x_star[] = { 0.066707422952301,  0.308024162523591, -0.843805757764051,
                     -1.276669375807300,  0.577067691426442};

  // Transfer variables to device.
  real_t *val_d, *b_d, *x_d;
  int *cind_d, *rptr_d;

  hipMalloc(&val_d, (nnz + m + n) * sizeof(real_t));
  hipMalloc(&cind_d, (nnz + m + 1) * sizeof(int));
  b_d = val_d + nnz;
  x_d = b_d + m;
  rptr_d = cind_d + nnz;

  hipMemcpy(val_d, val_h, nnz * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, m * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(x_d, x_h, n * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(cind_d, cind_h, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(rptr_d, rptr_h, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

  // Solve.
  int flag = cgls::solve<real_t, cgls::CSR>(val_d, rptr_d, cind_d, m, n,
      nnz, b_d, x_d, shift, tol, maxit, quiet);

  // Retrieve solution.
  hipMemcpy(x_h, x_d, n * sizeof(real_t), hipMemcpyDeviceToHost);

  // Compute error and print.
  real_t err = 0;
  for (int i = 0; i < n; ++i)
    err += (x_h[i] - x_star[i]) * (x_h[i] - x_star[i]);
  err = std::sqrt(err);
  if (flag == 0 && err < tol)
    printf("Test2 Passed: Flag = %d, Error = %e\n", flag, err);
  else
    printf("Test2 Failed: Flag = %d, Error = %e\n", flag, err);

  // Free data.
  hipFree(val_d);
  hipFree(cind_d);
}

// Test CGLS on larger random matrix.
void test3() {
  // Initialize variables.
  real_t shift = 1;
  real_t tol = 1e-6;
  int maxit = 100;
  bool quiet = false;
  int m = 100;
  int n = 1000;
  int nnz = 10000;

  // Initialize data.
  real_t *val_h = new real_t[nnz];
  int *cind_h = new int[nnz];
  int *rptr_h = new int[m + 1];
  real_t *b_h = new real_t[m];
  real_t *x_h = new real_t[n]();

  // Generate data.
  CsrMatGen(m, n, nnz, val_h, rptr_h, cind_h);
  for (int i = 0; i < m; ++i)
    b_h[i] = rand() / static_cast<real_t>(RAND_MAX);

  // Transfer variables to device.
  real_t *val_a_d, *b_d, *x_d;
  int *cind_a_d, *rptr_a_d;

  hipMalloc(&val_a_d, nnz * sizeof(real_t));
  hipMalloc(&x_d, n * sizeof(real_t));
  hipMalloc(&b_d, m * sizeof(real_t));
  hipMalloc(&cind_a_d, nnz * sizeof(int));
  hipMalloc(&rptr_a_d, (m + 1) * sizeof(int));

  hipMemcpy(val_a_d, val_h, nnz * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, m * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(x_d, x_h, n * sizeof(real_t), hipMemcpyHostToDevice);
  hipMemcpy(cind_a_d, cind_h, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(rptr_a_d, rptr_h, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

  // Make A^T copy.
  real_t *val_at_d;
  int *cind_at_d, *rptr_at_d;

  hipMalloc(&val_at_d, nnz * sizeof(real_t));
  hipMalloc(&cind_at_d, nnz * sizeof(int));
  hipMalloc(&rptr_at_d, (n + 1) * sizeof(int));
  rptr_at_d = cind_at_d + nnz;

  hipsparseHandle_t handle_s;
  hipsparseCreate(&handle_s);
  csr2csc(handle_s, m, n, nnz, val_a_d, rptr_a_d, cind_a_d, val_at_d,
      cind_at_d, rptr_at_d, HIPSPARSE_ACTION_NUMERIC,
      HIPSPARSE_INDEX_BASE_ZERO);
  hipDeviceSynchronize();
  hipsparseDestroy(handle_s);

  // Solve.
  int flag = cgls::solve<real_t, cgls::CSR>(val_a_d, rptr_a_d, cind_a_d,
      val_at_d, rptr_at_d, cind_at_d, m, n, nnz, b_d, x_d, shift, tol, maxit,
      quiet);

  // Check Result
  if (flag == 0)
    printf("Test3 Passed: Flag = %d\n", flag);
  else
    printf("Test3 Failed: Flag = %d\n", flag);

  // Free data.
  hipFree(val_a_d);
  hipFree(cind_a_d);
  hipFree(val_at_d);
  hipFree(cind_at_d);
  delete [] val_h;
  delete [] rptr_h;
  delete [] cind_h;
  delete [] x_h;
  delete [] b_h;
}

// Run tests.
int main() {
  test1();
  test2();
  test3();
}

